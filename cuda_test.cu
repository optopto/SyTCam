#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
//#include "mainwindow.h"
//#include "ui_mainwindow.h"

#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/viz/types.hpp>


#include <EmergentCameraAPIs.h>
#include <emergentframe.h>
#include <EvtParamAttribute.h>
#include <gigevisiondeviceinfo.h>
#include <EmergentFrameSave.h>
#include <EmergentAVIFile.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hipfft/hipfftw.h>
#include <hip/hip_runtime.h>


using namespace std;
using namespace cv;
using namespace Emergent;

extern "C" int callAdd();
//extern "C" void testTwo();
extern "C" void getFrameCam(uchar *Frame,uchar *Frame_2,uchar *Frame_3,uchar *Frame_4 ,int buffer);
extern "C" void testCuda_(CEmergentCamera *camera, CEmergentFrame *evtFrame);

#define BUF 616*800
#define N 20


/*__host__ void testParam(){

    void *data;
    unsigned int flag = 1;
    hipDeviceptr_t pointer = 0;
    hipError_t response;
    response = hipPointerSetAttribute(&flag,HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,pointer);



    uint64_t buf_id = 0;
    response = hipPointerGetAttribute(&buf_id, HIP_POINTER_ATTRIBUTE_BUFFER_ID, pointer);

}*/

__global__ void cudatxt(uchar *frame, uchar *frame_2, uchar *frame_3 ,uchar *frame_4, uchar *imagen){
    int j = threadIdx.x + blockIdx.x * blockDim.x;
if(j < BUF){
    imagen[j] = frame[j]+frame_2[j]+frame_3[j]+frame_4[j];
}

}

__global__ void getFrames(CEmergentCamera *cam, CEmergentFrame *evtF, uchar *imagen){
#ifndef __CUDA_ARCH__

    int j = threadIdx.x + blockIdx.x * blockDim.x;

   // if(j == 0){
    //   EVT_CameraGetFrame(cam,evtF,EVT_INFINITE);

    //}

    if(j < BUF){
    imagen[j] = evtF->imagePtr[j];
    }


#else

//comentario
#endif

}


void getFrameCam(uchar *Frame,uchar *Frame_2,uchar *Frame_3,uchar *Frame_4, int buffer){
    //srand (time(NULL));

    Mat mat;
    uchar * dev_frame;
    uchar * dev_frame_2;
    uchar * dev_frame_3;
    uchar * dev_frame_4;
    uchar * dev_img;
    uchar imagen[616][800];
//    uchar ranIm[616][800];

  /*  for(int i = 0; i < 616; i++){
        for(int j = 0; j<800;j++){
            ranIm[i][j] =  rand() % 2;
        }
    }*/

 /*   FILE *saveOriginal;
    saveOriginal = fopen("/home/pc/Documentos/matlab_/cuda_original.txt","w");
    for(int i = 0; i < 616; i++){
        for(int j = 0; j<800;j++)
            //       imagen2D[i][j] = Frame[j+(800*i)];
            fprintf(saveOriginal,"%d ", Frame[j+(800*i)]);
    }

    fclose(saveOriginal);*/

    hipMalloc((void**)&dev_frame, buffer*sizeof(uchar));
    hipMalloc((void**)&dev_frame_2, buffer*sizeof(uchar));
    hipMalloc((void**)&dev_frame_3, buffer*sizeof(uchar));
    hipMalloc((void**)&dev_frame_4, buffer*sizeof(uchar));
    hipMalloc((void**)&dev_img, buffer*sizeof(uchar));
    hipMemcpy(dev_frame,Frame,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    hipMemcpy(dev_frame_2,Frame_2,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    hipMemcpy(dev_frame_3,Frame_3,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    hipMemcpy(dev_frame_4,Frame_4,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    hipMemcpy(dev_img,imagen,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    cudatxt<<<800,616>>>(dev_frame, dev_frame_2, dev_frame_3, dev_frame_4, dev_img);
    hipMemcpy(imagen,dev_img,buffer*sizeof(uchar),hipMemcpyDeviceToHost);

   /* FILE *fileSave;
    fileSave = fopen("/home/pc/Documentos/matlab_/cuda.txt","w");
    for(int i = 0; i< 616; i++){
        for(int j = 0; j< 800; j++){
            fprintf(fileSave,"%d ", imagen[i][j]);
        }
    }

    fclose(fileSave);
*/
    /*mat = Mat(616, 800, CV_8UC1, imagen);
    namedWindow("cuda imagen", 1);
    imshow("cuda imagen",mat);
    waitKey(30);*/
    mat = Mat(616, 800, CV_8UC1, imagen);
    imwrite("/home/pc/Documentos/matlab_/sumaImagenes.tiff",mat);
   // imshow("procesada",mat);
   // waitKey(30)
    hipFree(dev_frame);
    hipFree(dev_frame_2);
    hipFree(dev_frame_3);
    hipFree(dev_frame_4);
    hipFree(dev_img);
    return;
}

void testCuda_(CEmergentCamera* camera, CEmergentFrame *evtFrame){
    cout << "1" << endl;

    CEmergentCamera *gpuCam;
    CEmergentFrame *gpuFrame;
    uchar * gpuImagen;
    uchar imagen[BUF];
    cout << "2" << endl;

    hipMalloc((void**)&gpuCam, sizeof(CEmergentCamera));
    hipMalloc((void**)&gpuFrame, sizeof(CEmergentFrame));
    hipMalloc((void**)&gpuImagen, BUF*sizeof(uchar));
    cout << "3" << endl;

    hipMemcpy(gpuCam,camera,sizeof(CEmergentCamera),hipMemcpyHostToDevice);
    hipMemcpy(gpuFrame,evtFrame,sizeof(CEmergentFrame),hipMemcpyHostToDevice);
    hipMemcpy(gpuImagen,imagen,BUF*sizeof(uchar),hipMemcpyHostToDevice);
    cout << "4" << endl;

    getFrames<<<800,616>>>(gpuCam,gpuFrame,gpuImagen);
    cout << "5" << endl;

    hipMemcpy(imagen,gpuImagen,BUF*sizeof(uchar),hipMemcpyDeviceToHost);
    cout << "6" << endl;

    FILE *save;
    save = fopen("/home/pc/Documentos/matlab_/prueba.txt","w");
    for(int  i = 0; i < BUF ; i++){
    //cout << (int)imagen[i] << endl;
    fprintf(save,"%d ", (int)imagen[i]);
    }
    fclose(save);
    cout << "7" << endl;

        Mat mat;
        mat = Mat(616, 800, CV_8UC1, imagen);
        imwrite("/media/pc/b889862b-2b8a-44b9-8595-8b3819d37d2b/imagenes/imagen_1.bmp",mat);


    hipFree(gpuCam);
    hipFree(gpuFrame);
    hipFree(gpuImagen);


}

/*void testTwo(){

    testParam<<<1,1>>>();
}*/
